#include "hip/hip_runtime.h"
/*
	CS 6023 Assignment 3. 
	Do not make any changes to the boiler plate code or the other files in the folder.
	Use hipFree to deallocate any memory not in usage.
	Optimize as much as possible.
 */

#include "SceneNode.h"
#include <queue>
#include "Renderer.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <chrono>


void readFile (const char *fileName, std::vector<SceneNode*> &scenes, std::vector<std::vector<int> > &edges, std::vector<std::vector<int> > &translations, int &frameSizeX, int &frameSizeY) {
	/* Function for parsing input file*/

	FILE *inputFile = NULL;
	// Read the file for input. 
	if ((inputFile = fopen (fileName, "r")) == NULL) {
		printf ("Failed at opening the file %s\n", fileName) ;
		return ;
	}

	// Input the header information.
	int numMeshes ;
	fscanf (inputFile, "%d", &numMeshes) ;
	fscanf (inputFile, "%d %d", &frameSizeX, &frameSizeY) ;
	

	// Input all meshes and store them inside a vector.
	int meshX, meshY ;
	int globalPositionX, globalPositionY; // top left corner of the matrix.
	int opacity ;
	int* currMesh ;
	for (int i=0; i<numMeshes; i++) {
		fscanf (inputFile, "%d %d", &meshX, &meshY) ;
		fscanf (inputFile, "%d %d", &globalPositionX, &globalPositionY) ;
		fscanf (inputFile, "%d", &opacity) ;
		currMesh = (int*) malloc (sizeof (int) * meshX * meshY) ;
		for (int j=0; j<meshX; j++) {
			for (int k=0; k<meshY; k++) {
				fscanf (inputFile, "%d", &currMesh[j*meshY+k]) ;
			}
		}
		//Create a Scene out of the mesh.
		SceneNode* scene = new SceneNode (i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity) ; 
		scenes.push_back (scene) ;
	}

	// Input all relations and store them in edges.
	int relations;
	fscanf (inputFile, "%d", &relations) ;
	int u, v ; 
	for (int i=0; i<relations; i++) {
		fscanf (inputFile, "%d %d", &u, &v) ;
		edges.push_back ({u,v}) ;
	}

	// Input all translations.
	int numTranslations ;
	fscanf (inputFile, "%d", &numTranslations) ;
	std::vector<int> command (3, 0) ;
	for (int i=0; i<numTranslations; i++) {
		fscanf (inputFile, "%d %d %d", &command[0], &command[1], &command[2]) ;
		translations.push_back (command) ;
	}
}


void writeFile (const char* outputFileName, int *hFinalPng, int frameSizeX, int frameSizeY) {
	/* Function for writing the final png into a file.*/
	FILE *outputFile = NULL; 
	if ((outputFile = fopen (outputFileName, "w")) == NULL) {
		printf ("Failed while opening output file\n") ;
	}
	
	for (int i=0; i<frameSizeX; i++) {
		for (int j=0; j<frameSizeY; j++) {
			fprintf (outputFile, "%d ", hFinalPng[i*frameSizeY+j]) ;
		}
		fprintf (outputFile, "\n") ;
	}
}

__global__ void applyTranformations(int *preOrder, int *size, int *nodeIndex, int **translations, int translationCount, int nodeCount, int * globalPosX, int *globalPosY ){
      int id = blockIdx.x*blockDim.x+ threadIdx.x;
      if(id>=translationCount)return;


      int prNode = translations[id][0];
      int moves[4][2] = {{-1,0}, {1, 0}, {0,-1}, {0,1}};

      int prNodeIdx = nodeIndex[prNode];
      for(int i=0; i<size[prNode]; i++){
        int childNode = preOrder[prNodeIdx + i];
        atomicAdd(&globalPosX[childNode], moves[translations[id][1]][0] *translations[id][2]);
        atomicAdd(&globalPosY[childNode], moves[translations[id][1]][1] *translations[id][2]);
      }

}




__global__ void generateOpacities(int **g_mesh,int * g_frameX,int * g_frameY,int * g_globalCoordinatesX,int * g_globalCoordinatesY,int *g_globalOp, int *gFinalPng, int *g_opacity,int  nodeCount, int frameSizeX, int frameSizeY){
      int id = blockIdx.x*blockDim.x+ threadIdx.x;
      if(id>=nodeCount)return;

      int startX = g_globalCoordinatesX[id];
      int endX = startX + g_frameX[id] ;
  
      int startY = g_globalCoordinatesY[id];
      int endY = startY + g_frameY[id] ;

      for(int i=startX; i<endX; i++){
        for(int j=startY; j<endY; j++){
          if(i<0 || j<0 || i>=frameSizeX || j>=frameSizeY)continue;

          int pos =  i*frameSizeY+j;
          atomicMax(&g_globalOp[pos], g_opacity[id]);
        }
      }
}
 
__global__ void generateFinalMatrix(int *nodeOpacityMap,int **g_mesh,int * g_frameX,int * g_frameY,int * g_globalCoordinatesX,int * g_globalCoordinatesY,int *g_globalOp, int *gFinalPng, int *g_opacity,int  nodeCount, int frameSizeX, int frameSizeY){
      int id = blockIdx.x*blockDim.x+ threadIdx.x;
      if(id>=frameSizeX * frameSizeY)return;
      int posX = id/frameSizeY;
      int posY = id%frameSizeY;

      if(g_globalOp[id] == -1){
        gFinalPng[id] = 0;
        return;
      }
      int mshId = nodeOpacityMap[g_globalOp[id]];


      int startX = g_globalCoordinatesX[mshId];
      int startY = g_globalCoordinatesY[mshId];

      int mCurrX = posX - startX;
      int mCurrY = posY - startY;

      int mshSizeY = g_frameY[mshId];
      int mshPos = mCurrX * mshSizeY + mCurrY;
      gFinalPng[id] = g_mesh[mshId][mshPos];
}

int generatePreOrder(int nodeCount, int *preOrder, int *size, int *nodeIndex, int &currCount, int *hOffset, int *hCsr, bool *visited, int currNode){
  preOrder[currCount] = currNode;
  nodeIndex[currNode] = currCount;
  visited[currNode] = true;
  int childCount = 0;
  currCount++;
  for(int i=hOffset[currNode]; i<hOffset[currNode+1]; i++){
     int nxtNode = hCsr[i];
     if(visited[nxtNode])continue;
     childCount+= generatePreOrder(nodeCount, preOrder, size, nodeIndex, currCount, hOffset, hCsr, visited, nxtNode);
  }
  childCount++;
  size[currNode] = childCount;
  return childCount;
}  


int main (int argc, char **argv) {
	
	// Read the scenes into memory from File.
	const char *inputFileName = argv[1] ;
	int* hFinalPng ; 

	int frameSizeX, frameSizeY ;
	std::vector<SceneNode*> scenes ;
	std::vector<std::vector<int> > edges ;
	std::vector<std::vector<int> > translations ;
	readFile (inputFileName, scenes, edges, translations, frameSizeX, frameSizeY) ;
	hFinalPng = (int*) malloc (sizeof (int) * frameSizeX * frameSizeY) ;
	
	// Make the scene graph from the matrices.
    Renderer* scene = new Renderer(scenes, edges) ;

	// Basic information.
	int V = scenes.size () ;
	int E = edges.size () ;
	int numTranslations = translations.size () ;

	// Convert the scene graph into a csr.
	scene->make_csr () ; // Returns the Compressed Sparse Row representation for the graph.
	int *hOffset = scene->get_h_offset () ;  
	int *hCsr = scene->get_h_csr () ;
	int *hOpacity = scene->get_opacity () ; // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
	int **hMesh = scene->get_mesh_csr () ; // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
	int *hGlobalCoordinatesX = scene->getGlobalCoordinatesX () ; // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
	int *hGlobalCoordinatesY = scene->getGlobalCoordinatesY () ; // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
	int *hFrameSizeX = scene->getFrameSizeX () ; // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
	int *hFrameSizeY = scene->getFrameSizeY () ; // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.

	auto start = std::chrono::high_resolution_clock::now () ;


	// Code begins here.
	// Do not change anything above this comment.


  int nodeCount = scene->getNumNodes(); 
  int *preOrder = (int*) malloc (sizeof (int) *nodeCount) ;
  int *size = (int*) malloc (sizeof (int) *nodeCount) ;
  int *nodeIndex = (int*) malloc (sizeof (int) *nodeCount) ;
  bool *visited = (bool*) malloc (sizeof (bool) *nodeCount) ;
  int currCount = 0;

  
  generatePreOrder(nodeCount, preOrder, size, nodeIndex, currCount, hOffset, hCsr, visited, 0);


  //copy preorder to gpu
  int *g_preorder;
  hipMalloc(&g_preorder, (sizeof (int) * nodeCount) );
  hipMemcpy(g_preorder, preOrder, sizeof(int)*nodeCount, hipMemcpyHostToDevice);

  int *g_size;
  hipMalloc(&g_size, (sizeof (int) * nodeCount) );
  hipMemcpy(g_size, size, sizeof(int)*nodeCount, hipMemcpyHostToDevice);
 
  int *g_nodeIndex;
  hipMalloc(&g_nodeIndex, (sizeof (int) * nodeCount) );
  hipMemcpy(g_nodeIndex, nodeIndex, sizeof(int)*nodeCount, hipMemcpyHostToDevice);
  
  //Apply all the translations in parallel
  //copy translations to GPU
  int **g_translations;
  hipMalloc(&g_translations, numTranslations * sizeof(int *));
  for (int i = 0; i < numTranslations; i++) {
      int *temp;
      hipMalloc(&temp, 3 * sizeof(int));
      hipMemcpy(temp, translations[i].data(), 3 * sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(g_translations+i, &temp, sizeof(int*), hipMemcpyHostToDevice);
  }

  //copy global coordinates
  int *g_globalCoordinatesX, *g_globalCoordinatesY;
  hipMalloc(&g_globalCoordinatesX, nodeCount*sizeof(int));
  hipMemcpy(g_globalCoordinatesX, hGlobalCoordinatesX, nodeCount * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&g_globalCoordinatesY, nodeCount*sizeof(int));
  hipMemcpy(g_globalCoordinatesY, hGlobalCoordinatesY, nodeCount * sizeof(int), hipMemcpyHostToDevice);


  int threadsPerBlock = 1024;
  int gpuBlocks = (numTranslations+threadsPerBlock-1)/threadsPerBlock;
  applyTranformations<<<gpuBlocks, threadsPerBlock>>>(g_preorder, g_size, g_nodeIndex, g_translations, numTranslations, nodeCount, g_globalCoordinatesX,g_globalCoordinatesY);  
  hipDeviceSynchronize();
  //Finally generate the scene matrix in parallel
  // copy opacity to gpu
  int *g_opacity;
  hipMalloc(&g_opacity, nodeCount*sizeof(int));
  hipMemcpy(g_opacity, hOpacity, nodeCount * sizeof(int), hipMemcpyHostToDevice);

  //generate and copy global opacity matrix gpu
  int *g_globalOp;
  hipMalloc(&g_globalOp, (sizeof (int) * frameSizeX * frameSizeY) );
  hipMemset(g_globalOp, -1,(sizeof (int) * frameSizeX * frameSizeY));


  //create final matrix in gpu
  int *gFinalPng;
  hipMalloc(&gFinalPng, (sizeof (int) * frameSizeX * frameSizeY));
  hipMemcpy(gFinalPng, hFinalPng, sizeof (int) * frameSizeX * frameSizeY, hipMemcpyHostToDevice);

  //create and copy frame sizes to GPU
  int *g_frameX, *g_frameY;
  hipMalloc(&g_frameX, nodeCount*sizeof(int));
  hipMemcpy(g_frameX, hFrameSizeX, nodeCount * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc(&g_frameY, nodeCount*sizeof(int));
  hipMemcpy(g_frameY, hFrameSizeY, nodeCount * sizeof(int), hipMemcpyHostToDevice);

  //copy hMesh to gpu
  int **g_mesh;
  hipMalloc(&g_mesh, nodeCount*sizeof(int *));
  for (int i = 0; i < nodeCount; i++) {
      int *temp;
      hipMalloc(&temp, hFrameSizeX[i]*hFrameSizeY[i]* sizeof(int));
      hipMemcpy(temp, hMesh[i], hFrameSizeX[i]*hFrameSizeY[i] * sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(g_mesh+i, &temp, sizeof(int*), hipMemcpyHostToDevice);
  }

 
  //generate final matrix
  gpuBlocks = (nodeCount+1023)/1024;
  generateOpacities<<<gpuBlocks, 1024>>>(g_mesh, g_frameX, g_frameY, g_globalCoordinatesX, g_globalCoordinatesY,g_globalOp,gFinalPng, g_opacity, nodeCount, frameSizeX, frameSizeY);
  // generateFinalMatrix<<<1, 1>>>(g_mesh, g_frameX, g_frameY, g_globalCoordinatesX, g_globalCoordinatesY,g_globalLock,g_globalOp,gFinalPng, g_opacity, nodeCount, frameSizeX, frameSizeY);

  int maxOpacity = 0;
  for(int i=0;i<nodeCount; i++){
    maxOpacity = max(maxOpacity, hOpacity[i]);
  }

  //generate a map for opacity & node mapping
  int *nodeOpacityMap = (int *)malloc(sizeof(int)*(maxOpacity + 10));

  for(int i=0;i<nodeCount; i++){
    int currOp = hOpacity[i];
    nodeOpacityMap[currOp] = i;
  }

  int *g_nodeOpacityMap ;
  hipMalloc(&g_nodeOpacityMap, sizeof(int)*(maxOpacity + 10));
  hipMemcpy(g_nodeOpacityMap, nodeOpacityMap, sizeof(int)*(maxOpacity + 10), hipMemcpyHostToDevice);


  int totalCells = frameSizeX * frameSizeY;
  gpuBlocks = (totalCells + threadsPerBlock -1)/threadsPerBlock;
  generateFinalMatrix<<<gpuBlocks, threadsPerBlock>>>(g_nodeOpacityMap,g_mesh, g_frameX, g_frameY, g_globalCoordinatesX, g_globalCoordinatesY,g_globalOp,gFinalPng, g_opacity, nodeCount, frameSizeX, frameSizeY);

  hipMemcpy(hFinalPng, gFinalPng, sizeof (int) * frameSizeX * frameSizeY, hipMemcpyDeviceToHost);

  // for(int i=0;i<frameSizeX; i++){
  //   for(int j=0;j<frameSizeY; j++){
  //     printf("%d ", hFinalPng[i*frameSizeY+j]);
  //   }
  //   printf("\n");
  // }
  // freeing cuda memory
  hipFree(g_frameX);
  hipFree(g_frameY);
  hipFree(g_globalCoordinatesX);
  hipFree(g_globalCoordinatesY);
  hipFree(g_globalOp);
  hipFree(g_mesh);
  hipFree(g_opacity);
  hipFree(g_translations);
  hipFree(g_preorder);
  hipFree(g_size);
  hipFree(g_nodeIndex);
  hipFree(g_nodeOpacityMap);
	// Do not change anything below this comment.
	// Code ends here.

	auto end  = std::chrono::high_resolution_clock::now () ;

	std::chrono::duration<double, std::micro> timeTaken = end-start;

	printf ("execution time : %f\n", timeTaken) ;
	// Write output matrix to file.
	const char *outputFileName = argv[2] ;
	writeFile (outputFileName, hFinalPng, frameSizeX, frameSizeY) ;	

}
